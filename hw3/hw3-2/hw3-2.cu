#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

/* Constants */
const int INF = ((1 << 30) - 1);
const int V = 50010;

/* Global variables */
int n, m; // n: # of vertices, m: # of edges
int* dist;

//======================
#define DEV_NO 0
hipDeviceProp_t prop;

/* Convert index */
int convert_index(int i, int j, int n) {
    return i * n + j;
}

/* Read input */
void input(char* infile) {
    // Read # of vertices and # of edges
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    // Allocate pinned memory for dist
    hipError_t stat = hipHostMalloc((void**)&dist, sizeof(int) * n * n, hipHostMallocDefault);
    if(stat != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(stat) << std::endl;
        exit(-1);
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << i << " " << j << std::endl;
            std::cout << convert_index(i, j, n) << std::endl;
            if (i == j) {
                dist[convert_index(i, j, n)] = 0;
            } else {
                dist[convert_index(i, j, n)] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        dist[convert_index(pair[0], pair[1], n)] = pair[2];
    }
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            int idx = convert_index(i, j, n);
            if (dist[idx] >= INF) {
                dist[idx] = INF;
            }
        }
        fwrite(dist + i * n, sizeof(int), n, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) {
    return (a + b - 1) / b;
}

int main(int argc, char* argv[]) {
    input(argv[1]);
    int B = 512;

    // cudaGetDeviceProperties(&prop, DEV_NO);
    // printf("maxThreadsPerBlock = %d, sharedMemPerBlock = %d\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);

    // block_FW(B);
    output(argv[2]);
    return 0;
}