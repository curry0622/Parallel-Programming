
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLK_FAC 2

const int INF = ((1 << 30) - 1);
const int V = 50010;

int vtx_num, edge_num, mtx_size;
int* h_dist;

__constant__ int d_vtx_num, d_mtx_size, d_blk_fac;

__device__ __host__ int convert_index(int i, int j, int row_size) {
    return i * row_size + j;
}

/* Get ceil(a / b) */
int ceil(int a, int b) {
    return (a + b - 1) / b;
}

/* Read file input */
void input(char* infile) {
    // Read vertex num and edge num
    FILE* file = fopen(infile, "rb");
    fread(&vtx_num, sizeof(int), 1, file);
    fread(&edge_num, sizeof(int), 1, file);

    // Calculate matrix size
    mtx_size = ceil(vtx_num, BLK_FAC) * BLK_FAC;
    printf("vtx_num: %d\n", vtx_num);
    printf("blk_fac: %d\n", BLK_FAC);
    printf("mtx_size: %d\n", mtx_size);

    // Allocate memory for h_dist
    hipHostMalloc((void**)&h_dist, sizeof(int) * mtx_size * mtx_size, hipHostMallocDefault);

    // Initialize h_dist
    for (int i = 0; i < mtx_size; ++i) {
        for (int j = 0; j < mtx_size; ++j) {
            int idx = convert_index(i, j, mtx_size);
            if(i == j && i < vtx_num && j < vtx_num)
                h_dist[idx] = 0;
            else
                h_dist[idx] = INF;
        }
    }

    // Read edges
    int pair[3];
    for (int i = 0; i < edge_num; ++i) {
        fread(pair, sizeof(int), 3, file);
        int idx = convert_index(pair[0], pair[1], mtx_size);
        h_dist[idx] = pair[2];
    }
    fclose(file);
}

/* Write file output */
void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < vtx_num; ++i) {
        for (int j = 0; j < vtx_num; ++j) {
            int idx = convert_index(i, j, mtx_size);
            if(h_dist[idx] >= INF)
                h_dist[idx] = INF;
        }
        fwrite(h_dist + i * mtx_size, sizeof(int), vtx_num, outfile);
    }
    fclose(outfile);
}

void cal(int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int B = BLK_FAC;
    int block_end_x = block_start_x + block_width;
    int block_end_y = block_start_y + block_height;
    int end_k = (Round + 1) * B > vtx_num ? vtx_num : (Round + 1) * B;

    for (int b_i = block_start_x; b_i < block_end_x; ++b_i) {
        for (int b_j = block_start_y; b_j < block_end_y; ++b_j) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times
            int block_internal_start_x = b_i * B;
            int block_internal_end_x = (b_i + 1) * B;
            int block_internal_start_y = b_j * B;
            int block_internal_end_y = (b_j + 1) * B;
            if (block_internal_end_x > vtx_num) block_internal_end_x = vtx_num;
            if (block_internal_end_y > vtx_num) block_internal_end_y = vtx_num;

            for (int k = Round * B; k < end_k; ++k) {
                // To calculate original index of elements in the block (b_i, b_j)
                // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
                #pragma omp parallel for schedule(dynamic, 1)
                for (int i = block_internal_start_x; i < block_internal_end_x; ++i) {
                    for (int j = block_internal_start_y; j < block_internal_end_y; ++j) {
                        int i_k = convert_index(i, k, mtx_size);
                        int k_j = convert_index(k, j, mtx_size);
                        int i_j = convert_index(i, j, mtx_size);
                        if (h_dist[i_k] + h_dist[k_j] < h_dist[i_j]) {
                            h_dist[i_j] = h_dist[i_k] + h_dist[k_j];
                        }
                    }
                }
            }
        }
    }
}

/* Phase 1's kernel */
__global__ void phase1(int* d_dist, int r) {
    // Get index of thread
    int j = threadIdx.x; // col idx
    int i = threadIdx.y; // row idx
    int s_idx = convert_index(i, j, d_blk_fac);
    int h_idx = convert_index(i + r * d_blk_fac, j + r * d_blk_fac, d_mtx_size);

    // Copy data from global memory to shared memory
    extern __shared__ int s_mem[];
    s_mem[s_idx] = d_dist[h_idx];

    // Compute
    for(int k = 0; k < d_blk_fac; ++k) {
        __syncthreads();
        int i_k_dist = s_mem[convert_index(i, k, d_blk_fac)];
        int k_j_dist = s_mem[convert_index(k, j, d_blk_fac)];
        if (i_k_dist + k_j_dist < s_mem[s_idx]) {
            s_mem[s_idx] = i_k_dist + k_j_dist;
        }
    }

    // Copy data from shared memory to global memory
    d_dist[h_idx] = s_mem[s_idx];
}

/* Phase 2's kernel */
__global__ void phase2(int* d_dist, int r) {
    // Get index of thread
    int j = threadIdx.x; // col idx
    int i = threadIdx.y; // row idx
    int real_i, real_j;
    int s_idx = convert_index(i, j, d_blk_fac), h_idx;
    int blk_size = d_blk_fac * d_blk_fac;

    // Copy data from global memory to shared memory
    if(blockIdx.x == 0) {
        // Pivot row
        if(blockIdx.y < r) {
            // Left blks of pivot blk
            real_i = i + r * d_blk_fac;
            real_j = j + blockIdx.y * d_blk_fac;
        } else {
            // Right blks of pivot blk
            real_i = i + r * d_blk_fac;
            real_j = j + (blockIdx.y + 1) * d_blk_fac;
        }
    } else {
        // Pivot col
        if(blockIdx.y < r) {
            // Up blks of pivot blk
            real_i = i + blockIdx.y * d_blk_fac;
            real_j = j + r * d_blk_fac;
        } else {
            // Down blks of pivot blk
            real_i = i + (blockIdx.y + 1) * d_blk_fac;
            real_j = j + r * d_blk_fac;
        }
    }
    h_idx = convert_index(real_i, real_j, d_mtx_size);

    extern __shared__ int s_mem[];
    s_mem[s_idx] = d_dist[h_idx]; // curr blk
    s_mem[blk_size + s_idx] = d_dist[convert_index(i + r * d_blk_fac, j + r * d_blk_fac, d_mtx_size)]; // pivot blk

    // Compute
    for(int k = 0; k < d_blk_fac; ++k) {
        __syncthreads();
        int i_k_dist = s_mem[blk_size + convert_index(i, k, d_blk_fac)]; // element in pivot blk
        int k_j_dist = s_mem[convert_index(k, j, d_blk_fac)]; // element in curr blk
        if (i_k_dist + k_j_dist < s_mem[s_idx]) {
            s_mem[s_idx] = i_k_dist + k_j_dist;
        }
    }

    // Copy data from shared memory to global memory
    d_dist[h_idx] = s_mem[s_idx];
}

/* Phase 2 row kernel */
__global__ void phase2_row(int* d_dist, int r) {
    // Get index of thread
    int j = threadIdx.x; // col idx
    int i = threadIdx.y; // row idx
    int real_i, real_j;
    int s_idx = convert_index(i, j, d_blk_fac), h_idx;
    int blk_size = d_blk_fac * d_blk_fac;

    // Copy data from global memory to shared memory
    if(blockIdx.x < r) {
        // Left blks of pivot blk
        real_i = i + r * d_blk_fac;
        real_j = j + blockIdx.x * d_blk_fac;
    } else {
        // Right blks of pivot blk
        real_i = i + r * d_blk_fac;
        real_j = j + (blockIdx.x + 1) * d_blk_fac;
    }
    h_idx = convert_index(real_i, real_j, d_mtx_size);

    extern __shared__ int s_mem[];
    s_mem[s_idx] = d_dist[h_idx]; // curr blk
    s_mem[blk_size + s_idx] = d_dist[convert_index(i + r * d_blk_fac, j + r * d_blk_fac, d_mtx_size)]; // pivot blk

    // Compute
    for(int k = 0; k < d_blk_fac; ++k) {
        __syncthreads();
        int i_k_dist = s_mem[blk_size + convert_index(i, k, d_blk_fac)]; // element in pivot blk
        int k_j_dist = s_mem[convert_index(k, j, d_blk_fac)]; // element in curr blk
        if (i_k_dist + k_j_dist < s_mem[s_idx]) {
            s_mem[s_idx] = i_k_dist + k_j_dist;
        }
    }

    // Copy data from shared memory to global memory
    d_dist[h_idx] = s_mem[s_idx];
}

/* Phase 2 col kernel */
__global__ void phase2_col(int* d_dist, int r) {
    // Get index of thread
    int j = threadIdx.x; // col idx
    int i = threadIdx.y; // row idx
    int real_i, real_j;
    int s_idx = convert_index(i, j, d_blk_fac), h_idx;
    int blk_size = d_blk_fac * d_blk_fac;

    // Copy data from global memory to shared memory
    if(blockIdx.x < r) {
        // Up blks of pivot blk
        real_i = i + blockIdx.x * d_blk_fac;
        real_j = j + r * d_blk_fac;
    } else {
        // Down blks of pivot blk
        real_i = i + (blockIdx.x + 1) * d_blk_fac;
        real_j = j + r * d_blk_fac;
    }
    h_idx = convert_index(real_i, real_j, d_mtx_size);

    extern __shared__ int s_mem[];
    s_mem[s_idx] = d_dist[h_idx]; // curr blk
    s_mem[blk_size + s_idx] = d_dist[convert_index(i + r * d_blk_fac, j + r * d_blk_fac, d_mtx_size)]; // pivot blk

    // Compute
    for(int k = 0; k < d_blk_fac; ++k) {
        __syncthreads();
        int i_k_dist = s_mem[convert_index(i, k, d_blk_fac)]; // element in curr blk
        int k_j_dist = s_mem[blk_size + convert_index(k, j, d_blk_fac)]; // element in pivot blk
        if (i_k_dist + k_j_dist < s_mem[s_idx]) {
            s_mem[s_idx] = i_k_dist + k_j_dist;
        }
    }

    // Copy data from shared memory to global memory
    d_dist[h_idx] = s_mem[s_idx];
}

void block_FW(int* d_dist) {
    int round = ceil(vtx_num, BLK_FAC);
    int s_mem_size = BLK_FAC * BLK_FAC * sizeof(int);
    dim3 thds_per_blk(BLK_FAC, BLK_FAC);
    dim3 p2_blks_per_grid(2, round - 1); // 2: 1 for row, 1 for col; round - 1: # of (blks in row(or col) - pivot blk)

    for (int r = 0; r < round; ++r) {
        printf("Round %d\n", r);
        /* Phase 1*/
        // cal(r, r, r, 1, 1);
        hipMemcpy(d_dist, h_dist, sizeof(int) * mtx_size * mtx_size, hipMemcpyHostToDevice);
        phase1<<<1, thds_per_blk, s_mem_size>>>(d_dist, r);
        hipMemcpy(h_dist, d_dist, mtx_size * mtx_size * sizeof(int), hipMemcpyDeviceToHost);

        for(int i = 0; i < vtx_num; i++) {
            for(int j = 0; j < vtx_num; j++) {
                printf("%d, %d -> %d\n", i, j, h_dist[convert_index(i, j, mtx_size)]);
            }
        }

        /* Phase 2*/
        // cal(r, r, 0, 1, r);
        // cal(r, r, r + 1, 1, round - r - 1);
        // cal(r, 0, r, r, 1);
        // cal(r, r + 1, r, round - r - 1, 1);

        hipMemcpy(d_dist, h_dist, sizeof(int) * mtx_size * mtx_size, hipMemcpyHostToDevice);
        phase2_row<<<round - 1, thds_per_blk, 2 * s_mem_size>>>(d_dist, r);
        phase2_col<<<round - 1, thds_per_blk, 2 * s_mem_size>>>(d_dist, r);
        hipMemcpy(h_dist, d_dist, mtx_size * mtx_size * sizeof(int), hipMemcpyDeviceToHost);

        // // FILE* file = fopen("output0.txt", "a");
        for(int i = 0; i < vtx_num; i++) {
            for(int j = 0; j < vtx_num; j++) {
                printf("%d, %d -> %d\n", i, j, h_dist[convert_index(i, j, mtx_size)]);
                // fprintf(file, "%d, %d -> %d\n", i, j, h_dist[i * n + j]);
            }
        }
        // fclose(file);
        // break;

        /* Phase 3*/
        cal(r, 0, 0, r, r);
        cal(r, 0, r + 1, r, round - r - 1);
        cal(r, r + 1, 0, round - r - 1, r);
        cal(r, r + 1, r + 1, round - r - 1, round - r - 1);
    }
}

int main(int argc, char* argv[]) {
    // Read input
    printf("Reading input...\n");
    input(argv[1]);
    printf("Read input done.\n");

    // Allocate memory for constants
    printf("Allocating memory for constants...\n");
    int blk_fac = BLK_FAC;
    hipMemcpyToSymbol(HIP_SYMBOL(d_vtx_num), &vtx_num, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_mtx_size), &mtx_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_blk_fac), &blk_fac, sizeof(int));
    printf("Allocate memory for constants done.\n");

    // Allocate memory for d_dist
    printf("Allocating memory...\n");
    int* d_dist;
    hipMalloc((void**)&d_dist, sizeof(int) * mtx_size * mtx_size);
    printf("Allocate memory done.\n");

    // Copy data from host to device
    printf("Copying data...\n");
    hipMemcpy(d_dist, h_dist, sizeof(int) * mtx_size * mtx_size, hipMemcpyHostToDevice);
    printf("Copy data done.\n");

    // Block FW
    printf("Block FW...\n");
    block_FW(d_dist);
    printf("Block FW done.\n");

    // Write output
    printf("Writing output...\n");
    output(argv[2]);
    printf("Write output done.\n");
    return 0;
}
