
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>

__global__ void add(int *a, int *b, int *c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    int a[2] = {130, 120}, b[2] = {210, 290}, c[2] = {0, 0}; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, 2 * sizeof(int));
    hipMalloc((void **)&d_b, 2 * sizeof(int));
    hipMalloc((void **)&d_c, 2 * sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, a, 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, 2 * sizeof(int), hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<1,1024>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, 2 * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << c[0] << " " << c[1] << std::endl;

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}