
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int INF = ((1 << 30) - 1);
const int V = 50010;

int n, m; // n: # of vertices, m: # of edges
int* h_dist;

__device__ __host__ int convert_index(int i, int j, int row_size) {
    return i * row_size + j;
}

/* Read file input */
void input(char* infile) {
    // Read n and m
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    // Allocate memory for h_dist
    hipHostMalloc((void**)&h_dist, sizeof(int) * n * n, hipHostMallocDefault);

    // Initialize h_dist
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            int idx = convert_index(i, j, n);
            if (i == j) {
                h_dist[idx] = 0;
            } else {
                h_dist[idx] = INF;
            }
        }
    }

    // Read edges
    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        int idx = convert_index(pair[0], pair[1], n);
        h_dist[idx] = pair[2];
    }
    fclose(file);
}

/* Write file output */
void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            int idx = convert_index(i, j, n);
            if(h_dist[idx] >= INF)
                h_dist[idx] = INF;
        }
        fwrite(h_dist + i * n, sizeof(int), n, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) {
    return (a + b - 1) / b;
}

void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int block_end_x = block_start_x + block_width;
    int block_end_y = block_start_y + block_height;
    int end_k = (Round + 1) * B > n ? n : (Round + 1) * B;

    for (int b_i = block_start_x; b_i < block_end_x; ++b_i) {
        for (int b_j = block_start_y; b_j < block_end_y; ++b_j) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times
            int block_internal_start_x = b_i * B;
            int block_internal_end_x = (b_i + 1) * B;
            int block_internal_start_y = b_j * B;
            int block_internal_end_y = (b_j + 1) * B;
            if (block_internal_end_x > n) block_internal_end_x = n;
            if (block_internal_end_y > n) block_internal_end_y = n;

            for (int k = Round * B; k < end_k; ++k) {
                // To calculate original index of elements in the block (b_i, b_j)
                // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
                #pragma omp parallel for schedule(dynamic, 1)
                for (int i = block_internal_start_x; i < block_internal_end_x; ++i) {
                    for (int j = block_internal_start_y; j < block_internal_end_y; ++j) {
                        int i_k = convert_index(i, k, n);
                        int k_j = convert_index(k, j, n);
                        int i_j = convert_index(i, j, n);
                        if (h_dist[i_k] + h_dist[k_j] < h_dist[i_j]) {
                            h_dist[i_j] = h_dist[i_k] + h_dist[k_j];
                        }
                    }
                }
            }
        }
    }
}

/* Phase 1's kernel */
extern __shared__ int s_dist[];
__global__ void phase1(int* d_dist, int B, int r, int n) {
    // Get index of thread
    int i = threadIdx.x;
    int j = threadIdx.y;
    int s_idx = convert_index(i, j, B);
    int h_idx = convert_index(i + r * B, j + r * B, n);

    // Copy data from global memory to shared memory
    s_dist[s_idx] = d_dist[h_idx];

    // Compute
    for(int k = 0; k < B; ++k) {
        __syncthreads();
        int i_k_dist = s_dist[convert_index(i, k, B)];
        int k_j_dist = s_dist[convert_index(k, j, B)];
        if (i_k_dist + k_j_dist < s_dist[s_idx]) {
            s_dist[s_idx] = i_k_dist + k_j_dist;
        }
    }

    // Copy data from shared memory to global memory
    d_dist[h_idx] = s_dist[s_idx];
}

void block_FW(int B, int* d_dist) {
    int round = ceil(n, B);
    for (int r = 0; r < round; ++r) {
        printf("Round %d\n", r);
        /* Phase 1*/
        // cal(B, r, r, r, 1, 1);
        // cudaMemcpy(d_dist, h_dist, sizeof(int) * n * n, cudaMemcpyHostToDevice);
        // dim3 thds_per_blk(B, B);
        // phase1<<<1, thds_per_blk, B * B * sizeof(int)>>>(d_dist, B, r, n);
        // cudaMemcpy(h_dist, d_dist, n * n * sizeof(int), cudaMemcpyDeviceToHost);

        // FILE* file = fopen("output0.txt", "a");
        for(int i = 0; i < n; i++) {
            for(int j = 0; j < n; j++) {
                printf("%d, %d -> %d\n", i, j, h_dist[i * n + j]);
                // fprintf(file, "%d, %d -> %d\n", i, j, h_dist[i * n + j]);
            }
        }
        // fclose(file);

        /* Phase 2*/
        cal(B, r, r, 0, 1, r);
        cal(B, r, r, r + 1, 1, round - r - 1);
        cal(B, r, 0, r, r, 1);
        cal(B, r, r + 1, r, round - r - 1, 1);

        /* Phase 3*/
        cal(B, r, 0, 0, r, r);
        cal(B, r, 0, r + 1, r, round - r - 1);
        cal(B, r, r + 1, 0, round - r - 1, r);
        cal(B, r, r + 1, r + 1, round - r - 1, round - r - 1);
    }
}

int main(int argc, char* argv[]) {
    // Read input
    printf("Reading input...\n");
    input(argv[1]);
    printf("Read input done.\n");

    // Allocate memory for d_dist
    printf("Allocating memory...\n");
    int* d_dist;
    hipMalloc((void**)&d_dist, sizeof(int) * n * n);
    printf("Allocate memory done.\n");

    // Copy data from host to device
    printf("Copying data...\n");
    hipMemcpy(d_dist, h_dist, sizeof(int) * n * n, hipMemcpyHostToDevice);
    printf("Copy data done.\n");

    // Block FW
    printf("Block FW...\n");
    int B = 32;
    block_FW(B, d_dist);
    printf("Block FW done.\n");

    // Write output
    printf("Writing output...\n");
    output(argv[2]);
    printf("Write output done.\n");
    return 0;
}
