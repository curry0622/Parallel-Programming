
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLK_FAC 32

const int INF = ((1 << 30) - 1);
const int V = 50010;

int vtx_num, edge_num, mtx_size;
int* h_dist;

__constant__ int d_vtx_num, d_mtx_size, d_blk_fac;

__device__ __host__ int convert_index(int i, int j, int row_size) {
    return i * row_size + j;
}

/* Get ceil(a / b) */
int ceil(int a, int b) {
    return (a + b - 1) / b;
}

/* Read file input */
void input(char* infile) {
    // Read vertex num and edge num
    FILE* file = fopen(infile, "rb");
    fread(&vtx_num, sizeof(int), 1, file);
    fread(&edge_num, sizeof(int), 1, file);

    // Calculate matrix size
    mtx_size = ceil(vtx_num, BLK_FAC) * BLK_FAC;
    printf("vtx_num: %d\n", vtx_num);
    printf("blk_fac: %d\n", BLK_FAC);
    printf("mtx_size: %d\n", mtx_size);

    // Allocate memory for h_dist
    hipHostMalloc((void**)&h_dist, sizeof(int) * mtx_size * mtx_size, hipHostMallocDefault);

    // Initialize h_dist
    for (int i = 0; i < mtx_size; ++i) {
        for (int j = 0; j < mtx_size; ++j) {
            int idx = convert_index(i, j, mtx_size);
            if(i == j && i < vtx_num && j < vtx_num)
                h_dist[idx] = 0;
            else
                h_dist[idx] = INF;
        }
    }

    // Read edges
    int pair[3];
    for (int i = 0; i < edge_num; ++i) {
        fread(pair, sizeof(int), 3, file);
        int idx = convert_index(pair[0], pair[1], mtx_size);
        h_dist[idx] = pair[2];
    }
    fclose(file);
}

/* Write file output */
void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < vtx_num; ++i) {
        for (int j = 0; j < vtx_num; ++j) {
            int idx = convert_index(i, j, mtx_size);
            if(h_dist[idx] >= INF)
                h_dist[idx] = INF;
        }
        fwrite(h_dist + i * mtx_size, sizeof(int), vtx_num, outfile);
    }
    fclose(outfile);
}

void cal(int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int B = BLK_FAC;
    int block_end_x = block_start_x + block_width;
    int block_end_y = block_start_y + block_height;
    int end_k = (Round + 1) * B > vtx_num ? vtx_num : (Round + 1) * B;

    for (int b_i = block_start_x; b_i < block_end_x; ++b_i) {
        for (int b_j = block_start_y; b_j < block_end_y; ++b_j) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times
            int block_internal_start_x = b_i * B;
            int block_internal_end_x = (b_i + 1) * B;
            int block_internal_start_y = b_j * B;
            int block_internal_end_y = (b_j + 1) * B;
            if (block_internal_end_x > vtx_num) block_internal_end_x = vtx_num;
            if (block_internal_end_y > vtx_num) block_internal_end_y = vtx_num;

            for (int k = Round * B; k < end_k; ++k) {
                // To calculate original index of elements in the block (b_i, b_j)
                // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
                #pragma omp parallel for schedule(dynamic, 1)
                for (int i = block_internal_start_x; i < block_internal_end_x; ++i) {
                    for (int j = block_internal_start_y; j < block_internal_end_y; ++j) {
                        int i_k = convert_index(i, k, mtx_size);
                        int k_j = convert_index(k, j, mtx_size);
                        int i_j = convert_index(i, j, mtx_size);
                        if (h_dist[i_k] + h_dist[k_j] < h_dist[i_j]) {
                            h_dist[i_j] = h_dist[i_k] + h_dist[k_j];
                        }
                    }
                }
            }
        }
    }
}

/* Phase 1's kernel */
extern __shared__ int s_dist[];
__global__ void phase1(int* d_dist, int r) {
    // Get index of thread
    int i = threadIdx.x;
    int j = threadIdx.y;
    int s_idx = convert_index(i, j, d_blk_fac);
    int h_idx = convert_index(i + r * d_blk_fac, j + r * d_blk_fac, d_mtx_size);

    // Copy data from global memory to shared memory
    s_dist[s_idx] = d_dist[h_idx];

    // Compute
    for(int k = 0; k < d_blk_fac; ++k) {
        __syncthreads();
        int i_k_dist = s_dist[convert_index(i, k, d_blk_fac)];
        int k_j_dist = s_dist[convert_index(k, j, d_blk_fac)];
        if (i_k_dist + k_j_dist < s_dist[s_idx]) {
            s_dist[s_idx] = i_k_dist + k_j_dist;
        }
    }

    // Copy data from shared memory to global memory
    d_dist[h_idx] = s_dist[s_idx];
}

void block_FW(int* d_dist) {
    int round = ceil(vtx_num, BLK_FAC);
    for (int r = 0; r < round; ++r) {
        printf("Round %d\n", r);
        /* Phase 1*/
        // cal(r, r, r, 1, 1);
        hipMemcpy(d_dist, h_dist, sizeof(int) * mtx_size * mtx_size, hipMemcpyHostToDevice);
        dim3 thds_per_blk(BLK_FAC, BLK_FAC);
        phase1<<<1, thds_per_blk, BLK_FAC * BLK_FAC * sizeof(int)>>>(d_dist, r);
        hipMemcpy(h_dist, d_dist, mtx_size * mtx_size * sizeof(int), hipMemcpyDeviceToHost);

        // FILE* file = fopen("output0.txt", "a");
        for(int i = 0; i < vtx_num; i++) {
            for(int j = 0; j < vtx_num; j++) {
                printf("%d, %d -> %d\n", i, j, h_dist[convert_index(i, j, mtx_size)]);
                // fprintf(file, "%d, %d -> %d\n", i, j, h_dist[i * n + j]);
            }
        }
        // fclose(file);
        // break;

        /* Phase 2*/
        cal(r, r, 0, 1, r);
        cal(r, r, r + 1, 1, round - r - 1);
        cal(r, 0, r, r, 1);
        cal(r, r + 1, r, round - r - 1, 1);

        /* Phase 3*/
        cal(r, 0, 0, r, r);
        cal(r, 0, r + 1, r, round - r - 1);
        cal(r, r + 1, 0, round - r - 1, r);
        cal(r, r + 1, r + 1, round - r - 1, round - r - 1);
    }
}

int main(int argc, char* argv[]) {
    // Read input
    printf("Reading input...\n");
    input(argv[1]);
    printf("Read input done.\n");

    // Allocate memory for constants
    printf("Allocating memory for constants...\n");
    int blk_fac = BLK_FAC;
    hipMemcpyToSymbol(HIP_SYMBOL(d_vtx_num), &vtx_num, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_mtx_size), &mtx_size, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_blk_fac), &blk_fac, sizeof(int));
    printf("Allocate memory for constants done.\n");

    // Allocate memory for d_dist
    printf("Allocating memory...\n");
    int* d_dist;
    hipMalloc((void**)&d_dist, sizeof(int) * mtx_size * mtx_size);
    printf("Allocate memory done.\n");

    // Copy data from host to device
    printf("Copying data...\n");
    hipMemcpy(d_dist, h_dist, sizeof(int) * mtx_size * mtx_size, hipMemcpyHostToDevice);
    printf("Copy data done.\n");

    // Block FW
    printf("Block FW...\n");
    block_FW(d_dist);
    printf("Block FW done.\n");

    // Write output
    printf("Writing output...\n");
    output(argv[2]);
    printf("Write output done.\n");
    return 0;
}
