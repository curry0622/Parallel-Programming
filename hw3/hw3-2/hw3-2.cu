
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int INF = ((1 << 30) - 1);
const int V = 50010;

int n, m; // n: # of vertices, m: # of edges
int* h_dist;

int convert_index(int i, int j, int row_size) {
    return i * row_size + j;
}

/* Read file input */
void input(char* infile) {
    // Read n and m
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    // Allocate memory for h_dist
    hipHostMalloc((void**)&h_dist, sizeof(int) * n * n, hipHostMallocDefault);

    // Initialize h_dist
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            int idx = convert_index(i, j, n);
            if (i == j) {
                h_dist[idx] = 0;
            } else {
                h_dist[idx] = INF;
            }
        }
    }

    // Read edges
    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        int idx = convert_index(pair[0], pair[1], n);
        h_dist[idx] = pair[2];
    }
    fclose(file);
}

/* Write file output */
void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            int idx = convert_index(i, j, n);
            if(h_dist[idx] >= INF)
                h_dist[idx] = INF;
        }
        fwrite(h_dist + i * n, sizeof(int), n, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) {
    return (a + b - 1) / b;
}

void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height) {
    int block_end_x = block_start_x + block_width;
    int block_end_y = block_start_y + block_height;
    int end_k = (Round + 1) * B > n ? n : (Round + 1) * B;

    for (int b_i = block_start_x; b_i < block_end_x; ++b_i) {
        for (int b_j = block_start_y; b_j < block_end_y; ++b_j) {
            // To calculate B*B elements in the block (b_i, b_j)
            // For each block, it need to compute B times
            int block_internal_start_x = b_i * B;
            int block_internal_end_x = (b_i + 1) * B;
            int block_internal_start_y = b_j * B;
            int block_internal_end_y = (b_j + 1) * B;
            if (block_internal_end_x > n) block_internal_end_x = n;
            if (block_internal_end_y > n) block_internal_end_y = n;

            for (int k = Round * B; k < end_k; ++k) {
                // To calculate original index of elements in the block (b_i, b_j)
                // For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
                #pragma omp parallel for schedule(dynamic, 1)
                for (int i = block_internal_start_x; i < block_internal_end_x; ++i) {
                    for (int j = block_internal_start_y; j < block_internal_end_y; ++j) {
                        int i_k = convert_index(i, k, n);
                        int k_j = convert_index(k, j, n);
                        int i_j = convert_index(i, j, n);
                        if (h_dist[i_k] + h_dist[k_j] < h_dist[i_j]) {
                            h_dist[i_j] = h_dist[i_k] + h_dist[k_j];
                        }
                    }
                }
            }
        }
    }
}

void block_FW(int B) {
    int round = ceil(n, B);
    for (int r = 0; r < round; ++r) {
        fflush(stdout);
        /* Phase 1*/
        cal(B, r, r, r, 1, 1);

        /* Phase 2*/
        cal(B, r, r, 0, 1, r);
        cal(B, r, r, r + 1, 1, round - r - 1);
        cal(B, r, 0, r, r, 1);
        cal(B, r, r + 1, r, round - r - 1, 1);

        /* Phase 3*/
        cal(B, r, 0, 0, r, r);
        cal(B, r, 0, r + 1, r, round - r - 1);
        cal(B, r, r + 1, 0, round - r - 1, r);
        cal(B, r, r + 1, r + 1, round - r - 1, round - r - 1);
    }
}

int main(int argc, char* argv[]) {
    input(argv[1]);
    int B = 512;
    block_FW(B);
    output(argv[2]);
    return 0;
}
